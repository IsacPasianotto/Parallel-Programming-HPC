#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <omp.h>
#include <mpi.h>


#include <hip/hip_runtime.h>
#include <hipblas.h>
#define N_THREADS_PER_BLOCK 32

#include "../include/init.h"
#include "../include/debug.h"
#include "../include/column_gathering.h"
#include "../include/product.h"
#include "../include/stopwatch.h"



void assign_gpu_to_process(int rank)
{
  int n_gpus;
  hipGetDeviceCount(&n_gpus);
  hipSetDevice(rank % n_gpus);
}

void get_ready_on_gpu(double* A, double* C, double* d_A, double* d_C, long int N, long int local_size, int rank, int size, double* time_records, int* time_counter)
{
  hipMalloc((void **) &d_A, local_size * N * sizeof(double));
  hipMalloc((void **) &d_C, local_size * N * sizeof(double));
  hipMemcpy(d_A, A, local_size * N * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_C, C, local_size * N * sizeof(double), hipMemcpyHostToDevice);

  record_time(time_records, time_counter);  // -- , t_cuda_2
}

__global__ void cuda_copy_block_to_global_c(double* d_C, double* local_C_block, long int N, long int local_size, int* all_sizes, int size, int iter)
{
  long int index = iter * ((N % size) > 0 ? N / size + 1 : N / size);
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < local_size && j < all_sizes[iter])
  {
    d_C[index + i * N + j] = local_C_block[i * all_sizes[iter] + j];
  }
}

void compute_block_result_cuda(double* d_A, double* d_C, double* buffer, long int N, long int local_size, int* all_sizes, int size, int iter, double* time_records, int* time_counter)
{

  record_time(time_records, time_counter);  // --- ;  t_cuda_{7 + 7 * iter}

  double *device_C_block;
  double *device_B_buffer;
  hipMalloc((void **) &device_C_block, local_size * all_sizes[iter] * sizeof(double));
  hipMalloc((void **) &device_B_buffer, buffer_size * N * sizeof(double));

  hipMemcpy(device_B_buffer, buffer, buffer_size * N * sizeof(double), hipMemcpyHostToDevice);
  hipMemset(device_C_block, 0.0, local_size * all_sizes[iter] * sizeof(double));

  record_time(time_records, time_counter);  // --- ;  t_cuda_{8 + 7 * iter}

  // perform the product
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  double alpha = 1.0;
  double beta = 1.0;
  hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, local_size, all_sizes[iter], N, &alpha, d_A, N, device_B_buffer, all_sizes[iter], &beta, device_C_block, all_sizes[iter]);

  dim3 threads(N_THREADS_PER_BLOCK, N_THREADS_PER_BLOCK);
  dim3 blocks((local_size + N_THREADS_PER_BLOCK - 1) / N_THREADS_PER_BLOCK, (all_sizes[iter] + N_THREADS_PER_BLOCK - 1) / N_THREADS_PER_BLOCK);
  cuda_copy_block_to_global_c<<<blocks, threads>>>(d_C, device_C_block, N, local_size, all_sizes, size, iter);

  hipDeviceSynchronize();  // wait for the kernel to finish
  record_time(time_records, time_counter);  // --- ;  t_cuda_{9 + 7 * iter}
}


void free_gpu_memory_loop(double* device_C_block, double* device_B_buffer, hipblasHandle_t handle)
{
  hipblasDestroy(handle);
  hipFree(device_C_block);
  hipFree(device_B_buffer);
}

void free_gpu_memory(double* d_A, double* d_C)
{
  hipFree(d_A);
  hipFree(d_C);
}
